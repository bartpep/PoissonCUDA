#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <omp.h>

#include "jacobi.cuh"


__global__ void d_jacobi(double* d_matrix, double* d_matrix_new, double* d_f, int N, double dec,  double* dif_out) {
    int i = blockIdx.x*lockDim.x + threadIdx.x;
    int j = blockIdx.y*lockDim.y + threadIdx.y;
    int k = blockIdx.z*lockDim.z + threadIdx.z;

    if(i > 0 && i < N-1 && j > 0 && j < N-1 && k > 0 & k < N-1){
        int idx = i + j*N +k*N*N;
        d_matrix_new[idx] = dec * ( d_matrix[(i-1) + N * j + N*N*k] +
                                    d_matrix[(i+1) + N * j + N*N*k] +
                                    d_matrix[i + N * (j-1) + N*N*k] +
                                    d_matrix[i + N * (j+1) + N*N*k] +
                                    d_matrix[i + N * j + N*N*(k-1)] +
                                    d_matrix[i + N * j + N*N*(k+1)] +
                                    d_f[idx]);
        atomicAdd(dif_out,d_matrix_new[idx]-d_matrix[idx])*(d_matrix_new[idx]-d_matrix[idx]);
    }
    hipDeviceSynchronize();

    //Update d_matrix to be the new matrix
    d_matrix = d_matrix_new;
    hipDeviceSynchronize();

    cudacudaMemcpy(dif, dif_out, cudaDeviceToHost);
}



double*** jacobi(double ***matrix, double ***matrix_new, double ***f, int N, int iterations){
    printf("Starting sequential version\n");
    // Multiplication is easier than division
    double end_time, start_time = omp_get_wtime(); 


    //Allocate GPU memory: device matrixes
    size_t size = N*N*N*sizeof(double);
    double *d_matrix, *d_matrix_new, *d_f, *dif_out, *h_dif_out;
    
    hipMalloc((void**)&d_matrix,size);
    hipMalloc((void**)&d_f,size);
    hipMalloc((void**)&dif_out, sizeof(double));
    
    end_time = omp_get_wtime();
    printf("Initialization time: %.2f", omp_get_wtime()- start_time);
    
    // Copy matrix values from host to device
    hipMemcpy(d_matrix,matrix,size,hipMemcpyHostToDevice);
    hipMemcpy(d_matrix_new,matrix_new,size,hipMemcpyHostToDevice);
    hipMemcpy(d_f,f,size,hipMemcpyHostToDevice);

    printf("Copy time: %.2f",omp_get_wtime() + end_time - start_time);
    end_time = omp_get_wtime();
    
    int count = 0;
    double dif = 100;
    while(count < iterations && dif > 1e-5){
        // Run Jacobi simulation on GPUs
        d_jacobi<<<NB,TPB>>>(d_matrix,d_matrix_new,d_f,N,iterations,start_time);
        
        //Update the necessary variables 
        end_time = omp_get_wtime(); 
        count++;
        printf("Final rounds: %.4f seconds\n", end_time - start_time);
    }

    // Transfer Result matrix back to host
    cudacudaMemcpy(matrix, d_matrix, cudaDeviceToHost);
    return matrix;
}